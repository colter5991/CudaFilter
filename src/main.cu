#include "hip/hip_runtime.h"
#include <stdio.h>

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION

#include "stb_image.h"
#include "stb_image_write.h"

#define NUM_CHANNELS 3
#define WINDOW_SIZE 3

__device__ int index(int x, int y, int width, int height) {
    return (y * NUM_CHANNELS * width) + (x * NUM_CHANNELS);
}

__device__ int square(int a) {return a * a;}

struct Quadrant {
    int x_start;
    int x_end;
    int y_start;
    int y_end;
};;

__global__ void oilpaint(const uint8_t* input, uint8_t* output, int width, int height) {
    // Iterate through image
    for (int y = blockIdx.y * blockDim.y + threadIdx.y; y < height; y += blockDim.y * gridDim.y) { 
    for (int x = blockIdx.x * blockDim.x + threadIdx.x; x < width;  x += blockDim.x * gridDim.x) {

        // Define 4 overlapping quadrants around the center pixel
        Quadrant quadrants[4] = {
            {max(x - WINDOW_SIZE, 0), x,                               max(y - WINDOW_SIZE, 0), y},
            {max(x - WINDOW_SIZE, 0), x,                               y,                       min(height - 1, y + WINDOW_SIZE)},
            {x,                       min(width - 1, x + WINDOW_SIZE), max(y - WINDOW_SIZE, 0), y},
            {x,                       min(width - 1, x + WINDOW_SIZE), y,                       min(height - 1, y + WINDOW_SIZE)},
        };

        // Calculate mean variance and intensity for each quadrant of the image
        int min_variance = 0x7FFFFFFF;
        uint8_t min_red = 0;
        uint8_t min_green = 0;
        uint8_t min_blue = 0;

        // Iterate through the 4 quadrants
        for (int i = 0; i < 4; i++) {
            Quadrant* quadrant = &quadrants[i];
            int red_sum = 0;
            int green_sum = 0;
            int blue_sum = 0;

            // First, get the mean brightness
            int brightness = 0;
            for (int y_quad = quadrant->y_start; y_quad <= quadrant->y_end; y_quad++) {
            for (int x_quad = quadrant->x_start; x_quad <= quadrant->x_end; x_quad++) {
                int red = input[index(x_quad, y_quad, width, height)];
                int green = input[index(x_quad, y_quad, width, height) + 1];
                int blue = input[index(x_quad, y_quad, width, height) + 2];
                red_sum += red;
                green_sum += green;
                blue_sum += blue;

                brightness += max(max(red, green), blue);
            }}
            int mean_brightness = brightness / square(WINDOW_SIZE + 1);


            // Next get the variance
            int variance = 0;
            for (int y_quad = quadrant->y_start; y_quad <= quadrant->y_end; y_quad++) {
            for (int x_quad = quadrant->x_start; x_quad <= quadrant->x_end; x_quad++) {
                int red = input[index(x_quad, y_quad, width, height)];
                int green = input[index(x_quad, y_quad, width, height) + 1];
                int blue = input[index(x_quad, y_quad, width, height) + 2];
                variance += square(max(max(red, green), blue) - mean_brightness);
            }}

            // Update the color if this variance is lower
            if (variance < min_variance) {
                variance = min_variance;
                min_red = (red_sum / square(WINDOW_SIZE + 1));
                min_green = (green_sum / square(WINDOW_SIZE + 1));
                min_blue = (blue_sum / square(WINDOW_SIZE + 1));
            }
        }

        // Write output
        output[index(x, y, width, height) + 0] = min_red;
        output[index(x, y, width, height) + 1] = min_green;
        output[index(x, y, width, height) + 2] = min_blue;
    }}
}

int main(int argc, char** argv) {
    if (argc != 3) {
        printf("usage: %s <input picture> <output picture>\n", argv[0]);
        return 1;
    }

    // Load input image
    int width;
    int height;
    int channels;
    const uint8_t* input_image = (const uint8_t*)stbi_load(argv[1], &width, &height, &channels, NUM_CHANNELS);
    if (input_image == NULL) {
        printf("Could not load image \"%s\"\n", argv[1]);
    }

    // Allocate input and output buffers
    uint8_t* d_input_image;
    hipError_t error;
    error = hipMalloc(&d_input_image, width * height * NUM_CHANNELS);
    if (error != hipSuccess) {
        printf("Failed to allocate gpu buffer: %s\n", hipGetErrorString(error));
        return 1;
    }
    uint8_t* d_output_image;
    error = hipMalloc(&d_output_image, width * height * NUM_CHANNELS);
    if (error != hipSuccess) {
        printf("Failed to allocate gpu buffer: %s\n", hipGetErrorString(error));
        return 1;
    }

    // Copy input buffer to gpu
    error = hipMemcpy(d_input_image, input_image, width * height * NUM_CHANNELS, hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        printf("Failed to copy memory from host to device: %s\n", hipGetErrorString(error));
        return 1;
    }

    // Call gpu kernel
    dim3 grid(32,32,1);
    dim3 block(8,8,1);
    oilpaint<<<grid, block>>>(d_input_image, d_output_image, width, height);


    // Copy output memory to local buffer
    uint8_t* output_image = (uint8_t*)malloc(width * height * NUM_CHANNELS);
    error = hipMemcpy(output_image, d_output_image, width * height * NUM_CHANNELS, hipMemcpyDeviceToHost);
    if (error != hipSuccess) {
        printf("Failed to copy memory from device to host: %s\n", hipGetErrorString(error));
        return 1;
    }

    // Write output to image file
    int stbi_error = stbi_write_bmp(argv[2], width, height, NUM_CHANNELS, output_image);
    if (stbi_error == 0) {
        printf("Failed to write to output image \"%s\"\n", argv[2]);
        return 1;
    }

    return 0;

}

